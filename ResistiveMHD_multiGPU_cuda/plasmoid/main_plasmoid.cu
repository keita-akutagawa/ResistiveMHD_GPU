#include "hip/hip_runtime.h"
#include "main_plasmoid_const.hpp"


__global__ void initializeU_kernel(
    ConservationParameter* U, 
    MPIInfo* device_mPIInfo
) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < device_nx && j < device_ny) {
        MPIInfo mPIInfo = *device_mPIInfo;

        if (mPIInfo.isInside(i, j)) {
            int index = mPIInfo.globalToLocal(i, j);

            double rho, u, v, w, bX, bY, bZ, e, p;
            double bXHalf, bYHalf;
            double y = j * device_dy;
            double yCenter = 0.5 * (device_ymax - device_ymin);
            double xi, kmax;
            double coef = 5.0 * device_sheatThickness; 
            xi = (y - yCenter) / coef;
            kmax = 2.0 * device_PI / device_xmax;
            
            rho = device_rho0 * (sqrt(device_betaUpstream) + pow(cosh((y - yCenter) / device_sheatThickness), -2));
            u = -device_triggerRatio * device_VA / coef / kmax / pow(cosh((y - yCenter) / coef), 2) * sin(kmax * i * device_dx);
            v = device_triggerRatio * device_VA * tanh(xi) * cos(kmax * i * device_dx);
            w = 0.0;
            bX = device_b0 * tanh((y - yCenter) / device_sheatThickness);
            bXHalf = device_b0 * tanh((y - yCenter) / device_sheatThickness);
            bY = 0.0;
            bYHalf = 0.0;
            bZ = 0.0;
            p = device_p0 * (device_betaUpstream + pow(cosh((y - yCenter) / device_sheatThickness), -2));
            e = p / (device_gamma_mhd - 1.0)
            + 0.5 * rho * (u * u + v * v + w * w)
            + 0.5 * (bX * bX + bY * bY + bZ * bZ);
            
            U[index].rho  = rho;
            U[index].rhoU = rho * u;
            U[index].rhoV = rho * v;
            U[index].rhoW = rho * w;
            U[index].bX   = bXHalf;
            U[index].bY   = bYHalf;
            U[index].bZ   = bZ;
            U[index].e    = e;
        }
    }
}

void ResistiveMHD2D::initializeU()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_sheatThickness), &sheatThickness, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_betaUpstream), &betaUpstream, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_rho0), &rho0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_b0), &b0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_p0), &p0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_VA), &VA, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_alfvenTime), &alfvenTime, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_eta), &eta, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_triggerRatio), &triggerRatio, sizeof(double));

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeU_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        device_mPIInfo
    );
    hipDeviceSynchronize();

    sendrecv_U(U, mPIInfo);
    boundary.periodicBoundaryX2nd_U(U);
    boundary.wallBoundaryY2nd_U(U);
    MPI_Barrier(MPI_COMM_WORLD);
}


__device__
double getEta(double& xPosition, double& yPosition)
{
    double etaLocal;

    etaLocal = device_eta;
    
    return etaLocal;
}



//////////////////////////////////////////////////


int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    MPIInfo mPIInfo;
    setupInfo(mPIInfo, buffer);

    if (mPIInfo.rank == 0) {
        std::cout << mPIInfo.gridX << "," << mPIInfo.gridY << std::endl;
        logfile   << mPIInfo.gridX << "," << mPIInfo.gridY << std::endl;
    }

    hipSetDevice(mPIInfo.rank);

    initializeDeviceConstants();


    ResistiveMHD2D resistiveMHD2D(mPIInfo);
    MPI_Barrier(MPI_COMM_WORLD);

    resistiveMHD2D.initializeU();

    if (mPIInfo.rank == 0) {
        size_t free_mem = 0;
        size_t total_mem = 0;
        hipError_t status = hipMemGetInfo(&free_mem, &total_mem);

        std::cout << "Free memory: " << free_mem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "Total memory: " << total_mem / (1024 * 1024) << " MB" << std::endl;
    }

    for (step = 0; step < totalStep+1; step++) {
        MPI_Barrier(MPI_COMM_WORLD);

        hipMemcpyToSymbol(HIP_SYMBOL(device_totalTime), &totalTime, sizeof(double));

        if (step % recordStep == 0) {
            if (mPIInfo.rank == 0) {
                logfile << std::to_string(step) << ","
                        << std::setprecision(4) << totalTime
                        << std::endl;
                std::cout << std::to_string(step) << " step done : total time is "
                        << std::setprecision(4) << totalTime
                        << std::endl;
            }
            resistiveMHD2D.save(directoryname, filenameWithoutStep, step);
        }
        
        resistiveMHD2D.oneStepRK2_periodicXWallY();

        if (resistiveMHD2D.checkCalculationIsCrashed()) {
            std::cout << "Calculation stopped! : " << step << " steps" << std::endl;
            return 0;
        }

        if (mPIInfo.rank == 0) {
            totalTime += dt;
        }
    }
    
    MPI_Finalize();

    if (mPIInfo.rank == 0) {
        std::cout << "program was completed!" << std::endl;
    }

    return 0;
}


