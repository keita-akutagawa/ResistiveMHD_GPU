#include "const.hpp"


void initializeDeviceConstants() {
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dx), &dx, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin), &xmin, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax), &xmax, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_nx), &nx, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dy), &dy, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin), &ymin, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax), &ymax, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ny), &ny, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_CFL), &CFL, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(device_gamma_mhd), &gamma_mhd, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt), &dt, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(device_eta), &eta, sizeof(float));
}
