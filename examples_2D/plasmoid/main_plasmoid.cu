#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <string>
#include <iostream>
#include <fstream>
#include <iomanip>
#include "../../lib_ResistiveMHD_2D_GPU_symmetricXY/const.hpp"
#include "../../lib_ResistiveMHD_2D_GPU_symmetricXY/resistiveMHD_2D.hpp"


std::string directoryname = "results";
std::string filenameWithoutStep = "plasmoid";
std::ofstream logfile("results/log_plasmoid.txt");

const double EPS = 1e-20;
const double PI = 3.141592653589793;

const double gamma_mhd = 5.0 / 3.0;

const double sheat_thickness = 1.0;
const double betaUpstream = 0.2;
const double rho0 = 1.0;
const double b0 = 1.0;
const double p0 = b0 * b0 / 2.0;

const double eta0 = 1.0 / 500.0;
const double triggerRatio = 0.1;

const double xmin = 0.0;
const double xmax = 200.0;
const double dx = sheat_thickness / 20.0;
const int nx = int((xmax - xmin) / dx);
const double ymin = 0.0;
const double ymax = 20.0;
const double dy = sheat_thickness / 20.0;
const int ny = int((ymax - ymin) / dy);

const double CFL = 0.7;
double dt = 0.0;
const int totalStep = 50000;
const int recordStep = 100;
double totalTime = 0.0;

__constant__ double device_EPS;
__constant__ double device_PI;

__constant__ double device_dx;
__constant__ double device_xmin;
__constant__ double device_xmax;
__constant__ int device_nx;

__constant__ double device_dy;
__constant__ double device_ymin;
__constant__ double device_ymax;
__constant__ int device_ny;

__constant__ double device_CFL;
__constant__ double device_gamma_mhd;

__device__ double device_dt;

__constant__ double device_sheat_thickness;
__constant__ double device_betaUpstream;
__constant__ double device_rho0;
__constant__ double device_b0;
__constant__ double device_p0;

__constant__ double device_eta0;

__constant__ double device_triggerRatio;


__global__ void initializeU_kernel(ConservationParameter* U) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < device_nx - 1 && j < device_ny - 1) {
        double rho, u, v, w, bX, bY, bZ, e, p;
        double bXPlus1, bYPlus1; 
        double x = i * device_dx, y = j * device_dy;
        double xPlus1 = (i + 1) * device_dx, yPlus1 = (j + 1) * device_dy;
        
        rho = device_rho0 * (device_betaUpstream + pow(cosh((y - 0.5 * (device_ymax - device_ymin)) / device_sheat_thickness), -2));
        u = 0.0;
        v = 0.0;
        w = 0.0;
        bX = device_b0 * tanh((y - 0.5 * device_ymax) / device_sheat_thickness)
           - device_b0 * device_triggerRatio * (y - 0.5 * (device_ymax - device_ymin)) / device_sheat_thickness
           * exp(-(pow((x - 0.5 * (device_xmax - device_xmin)), 2) + pow((y - 0.5 * (device_ymax - device_ymin)), 2))
           / pow(2.0 * device_sheat_thickness, 2));
        bXPlus1 = device_b0 * tanh((y - 0.5 * device_ymax) / device_sheat_thickness)
                - device_b0 * device_triggerRatio * (y - 0.5 * (device_ymax - device_ymin)) / device_sheat_thickness
                * exp(-(pow((xPlus1 - 0.5 * (device_xmax - device_xmin)), 2) + pow((y - 0.5 * (device_ymax - device_ymin)), 2))
                / pow(2.0 * device_sheat_thickness, 2));
        bX = 0.5 * (bX + bXPlus1);
        bY = device_b0 * device_triggerRatio * (x - 0.5 * (device_xmax - device_xmin)) / device_sheat_thickness
           * exp(-(pow((x - 0.5 * (device_xmax - device_xmin)), 2) + pow((y - 0.5 * (device_ymax - device_ymin)), 2))
           / pow(2.0 * device_sheat_thickness, 2));
        bYPlus1 = device_b0 * device_triggerRatio * (x - 0.5 * (device_xmax - device_xmin)) / device_sheat_thickness
                * exp(-(pow((x - 0.5 * (device_xmax - device_xmin)), 2) + pow((yPlus1 - 0.5 * (device_ymax - device_ymin)), 2))
                / pow(2.0 * device_sheat_thickness, 2));
        bY = 0.5 * (bY + bYPlus1);
        bZ = 0.0;
        p = device_p0 * (device_betaUpstream + pow(cosh((y - 0.5 * (device_ymax - device_ymin)) / device_sheat_thickness), -2));
        e = p / (device_gamma_mhd - 1.0)
          + 0.5 * rho * (u * u + v * v + w * w)
          + 0.5 * (bX * bX + bY * bY + bZ * bZ);
        
        U[j + i * device_ny].rho  = rho;
        U[j + i * device_ny].rhoU = rho * u;
        U[j + i * device_ny].rhoV = rho * v;
        U[j + i * device_ny].rhoW = rho * w;
        U[j + i * device_ny].bX   = bX;
        U[j + i * device_ny].bY   = bY;
        U[j + i * device_ny].bZ   = bZ;
        U[j + i * device_ny].e    = e;
    }
}

void ResistiveMHD2D::initializeU()
{
    hipMemcpyToSymbol(HIP_SYMBOL(device_sheat_thickness), &sheat_thickness, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_betaUpstream), &betaUpstream, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_rho0), &rho0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_b0), &b0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_p0), &p0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_eta0), &eta0, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_triggerRatio), &triggerRatio, sizeof(double));

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    initializeU_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();

    boundary.symmetricBoundaryX2nd(U);
    boundary.symmetricBoundaryY2nd(U);
}


__device__
inline double getEta(double& xPosition, double& yPosition)
{
    double eta;

    eta = device_eta0;
    
    return eta;
}


__global__ void addResistiveTermToFluxF_kernel(
    const ConservationParameter* U, Flux* flux)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < device_nx - 2) && (0 < j) && (j < device_ny - 2)) {
        double xPosition = i * device_dx, yPosition = j * device_dy;
        double xPositionPlus1 = (i + 1) * device_dx;

        double jY, jZ;
        double eta;
        double etaJY, etaJYPlus1, etaJZ, etaJZPlus1;
        double etaJYBZ, etaJYBZPlus1, etaJZBY, etaJZBYPlus1;

        jY = -(U[j + (i + 1) * device_ny].bZ - U[j + (i - 1) * device_ny].bZ) / (2.0 * device_dx);
        jZ = (U[j + (i + 1) * device_ny].bY - U[j + (i - 1) * device_ny].bY) / (2.0 * device_dx)
           - (U[j + 1 + i * device_ny].bX - U[j - 1 + i * device_ny].bX) / (2.0 * device_dy);
        
        eta = getEta(xPosition, yPosition);
        etaJY = eta * jY; 
        etaJZ = eta * jZ;
        etaJYBZ = etaJY * U[j + i * device_ny].bZ;
        etaJZBY = etaJZ * U[j + i * device_ny].bY;

        jY = -(U[j + (i + 2) * device_ny].bZ - U[j + i * device_ny].bZ) / (2.0 * device_dx);
        jZ = (U[j + (i + 2) * device_ny].bY - U[j + i * device_ny].bY) / (2.0 * device_dx)
           - (U[j + 1 + (i + 1) * device_ny].bX - U[j - 1 + (i + 1) * device_ny].bX) / (2.0 * device_dy);
        
        eta = getEta(xPositionPlus1, yPosition);
        etaJYPlus1 = eta * jY; 
        etaJZPlus1 = eta * jZ;
        etaJYBZPlus1 = etaJY * U[j + (i + 1) * device_ny].bZ;
        etaJZBYPlus1 = etaJZ * U[j + (i + 1) * device_ny].bY;
  
        flux[j + i * device_ny].f5 -= 0.5 * (etaJZ + etaJZPlus1);
        flux[j + i * device_ny].f6 += 0.5 * (etaJY + etaJYPlus1);
        flux[j + i * device_ny].f7 += 0.5 * (etaJYBZ + etaJYBZPlus1)
                                    - 0.5 * (etaJZBY + etaJZBYPlus1);
    }
}

void FluxSolver::addResistiveTermToFluxF(
    const thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    addResistiveTermToFluxF_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(flux.data())
    );

    hipDeviceSynchronize();
}


__global__ void addResistiveTermToFluxG_kernel(
    const ConservationParameter* U, Flux* flux)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < device_nx - 2) && (0 < j) && (j < device_ny - 2)) {
        double xPosition = i * device_dx, yPosition = j * device_dy;
        double yPositionPlus1 = (j + 1) * device_dy;

        double jX, jZ;
        double eta;
        double etaJX, etaJXPlus1, etaJZ, etaJZPlus1;
        double etaJZBX, etaJZBXPlus1, etaJXBZ, etaJXBZPlus1;

        jX = (U[j + 1 + i * device_ny].bZ - U[j - 1 + i * device_ny].bZ) / (2.0 * device_dy);
        jZ = (U[j + (i + 1) * device_ny].bY - U[j + (i - 1) * device_ny].bY) / (2.0 * device_dx)
           - (U[j + 1 + i * device_ny].bX - U[j - 1 + i * device_ny].bX) / (2.0 * device_dy);
        
        eta = getEta(xPosition, yPosition);
        etaJX = eta * jX;
        etaJZ = eta * jZ;
        etaJXBZ = etaJX * U[j + i * device_ny].bZ;
        etaJZBX = etaJZ * U[j + i * device_ny].bX;

        jX = (U[j + 2 + i * device_ny].bZ - U[j + i * device_ny].bZ) / (2.0 * device_dy);
        jZ = (U[j + 1 + (i + 1) * device_ny].bY - U[j + 1 + (i - 1) * device_ny].bY) / (2.0 * device_dx)
           - (U[j + 2 + i * device_ny].bX - U[j + i * device_ny].bX) / (2.0 * device_dy);
        
        eta = getEta(xPosition, yPositionPlus1);
        etaJXPlus1 = eta * jX;
        etaJZPlus1 = eta * jZ;
        etaJXBZPlus1 = etaJX * U[j + 1 + i * device_ny].bZ;
        etaJZBXPlus1 = etaJZ * U[j + 1 + i * device_ny].bX;
  
        flux[j + i * device_ny].f4 += 0.5 * (etaJZ + etaJZPlus1);
        flux[j + i * device_ny].f6 -= 0.5 * (etaJX + etaJXPlus1);
        flux[j + i * device_ny].f7 += 0.5 * (etaJZBX + etaJZBXPlus1)
                                    - 0.5 * (etaJXBZ + etaJXBZPlus1);
    }
}

void FluxSolver::addResistiveTermToFluxG(
    const thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    addResistiveTermToFluxG_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(flux.data())
    );

    hipDeviceSynchronize();
}



//////////////////////////////////////////////////


int main()
{
    initializeDeviceConstants();

    ResistiveMHD2D resistiveMHD2D;

    resistiveMHD2D.initializeU();

    for (int step = 0; step < totalStep+1; step++) {
        if (step % recordStep == 0) {
            resistiveMHD2D.save(directoryname, filenameWithoutStep, step);
            logfile << std::to_string(step) << ","
                    << std::setprecision(4) << totalTime
                    << std::endl;
            std::cout << std::to_string(step) << " step done : total time is "
                      << std::setprecision(4) << totalTime
                      << std::endl;
        }
        
        resistiveMHD2D.oneStepRK2();

        if (resistiveMHD2D.checkCalculationIsCrashed()) {
            std::cout << "Calculation stopped! : " << step << " steps" << std::endl;
            return 0;
        }

        totalTime += dt;
    }
    
    return 0;
}


